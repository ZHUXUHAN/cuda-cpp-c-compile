#include "hip/hip_runtime.h"
// mathutil_cuda_kernel.cu
// 头文件，最后一个是cuda特有的
#include <hiprand.h>
#include <stdio.h>
#include <math.h>
#include <float.h>
#include "mathutil_cuda_kernel.h"

// 获取GPU线程通道信息
dim3 cuda_gridsize(int n)
{
    int k = (n - 1) / BLOCK + 1;
    int x = k;
    int y = 1;
    if(x > 65535) {
        x = ceil(sqrt(k));
        y = (n - 1) / (x * BLOCK) + 1;
    }
    dim3 d(x, y, 1);
    return d;
}
// 这个函数是cuda执行函数，可以看到细化到了每一个元素
__global__ void broadcast_sum_kernel(float *a, float *b, int x, int y, int size)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if(i >= size) return;
    int j = i % x; i = i / x;
    int k = i % y;
    a[IDX2D(j, k, y)] += b[k];
}


// 这个函数是与c语言函数链接的接口函数
void broadcast_sum_cuda(float *a, float *b, int x, int y, hipStream_t stream)
{
    int size = x * y;
    hipError_t err;

    // 上面定义的函数
    broadcast_sum_kernel<<<cuda_gridsize(size), BLOCK, 0, stream>>>(a, b, x, y, size);

    err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
//上面的代码和C语言很像，但是多了一个__global__,这个是cuda中特有类型，这个函数实现向量a和b的element-wise的相加。